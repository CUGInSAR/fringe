#include "hip/hip_runtime.h"
//
// Author: Piyush Agram
// Copyright 2018

#include "fringe/cuda/cudaUtils.h"
#include "fringe/cuda/ulongmask.h"
#include "KS2sample_cuda.h"
#include <math.h>
#include <iostream>

#define THRD_PER_BLOCK 96

//Constant memory for constant input values
//Done this way in topozero. Need to understand
//why this cannot be put in part of the struct
__constant__ double d_inpts_pval[1];
__constant__ int d_inpts_int[6];
//0: numCols
//1: numLines
//2: nBands
//3: Nx
//4: Ny
//5: wtslen

/*******Advance function declarations***********/
__global__ void runSortAmp(float *amp);
__global__ void findNeighbors(const float *amp, const unsigned char *mask,
                            unsigned int *wts, int * count);

/**********Inplace sorting algorithm************/
__device__ inline void bruteForceSort(float *arr)
{
    int ii,jj;
    float temp;

    for(jj=0; jj < (d_inpts_int[2]-1); jj++)
    {
        for (ii=0; ii < (d_inpts_int[2]-jj-1); ii++)
        {
            temp = fminf(arr[ii], arr[ii+1]);
            arr[ii+1] = fmaxf(arr[ii], arr[ii+1]);
            arr[ii] = temp;
        }
    }
};

/***** Structure for all GPU/Host handling data *****/
struct gpuParams
{
    //These are meant to be inputs
    //Pointers refer to pointers on device
    float *amplitude; //Flattened array of amplitude values
    unsigned char *mask; //Flattened mask array
    
    //These are host end variables
    int numCols;      //Number of cols in a line
    int numLines;     //Number of lines in a block
    int nBands;       //Number of bands
    int numWtsBands;    //Number of uint32 bands for nmap
    int Nx;           //Half window width in X 
    int Ny;           //Half window width in Y

    //These are meant to be outputs
    //Pointers refer to pointers on device
    int *count;         //Count of number of neighbors
    unsigned int *wts;  //Weights

    //Constructor and destructor
    gpuParams(int cols, int lines, int bands,
              int nx, int ny, int nwts);
    ~gpuParams();

    //Methods to help
    void allocateArrays();
    void setConstants();
    void deallocateArrays();
    void setInputs(float *, unsigned char *);
    void getOutputs(int *, unsigned int *);
    void sortAmplitude();
    void process(double pval);
};

gpuParams::gpuParams(int cols, int lines, int bands,
                     int nx, int ny, int nwts):
                     numCols(cols),numLines(lines),
                     nBands(bands),
                     Nx(nx), Ny(ny),
                     numWtsBands(nwts)
{
    //Ensure the memory on GPU is allocated for this
    allocateArrays();

    //Assign constants to the global arrays
    setConstants();
}

gpuParams::~gpuParams()
{
    //Ensure memory on GPU is released
    deallocateArrays();
}

//Allocate memory on the GPU
void gpuParams::allocateArrays()
{

    size_t nPix = numCols * numLines;

    //Allocate memory for input amplitude
    gpuErrChk ( hipMalloc((float**)&amplitude, (nBands*nPix)*sizeof(float)));
   gpuErrChk( hipMemset(amplitude, 0, (nBands*nPix)*sizeof(float)));

    //Allocate memory for input mask
    gpuErrChk( hipMalloc((unsigned char**)&mask, (nPix)*sizeof(unsigned char)));
    gpuErrChk( hipMemset(mask, 0, nPix*sizeof(unsigned char)));

    //Allocate memory for output count
    gpuErrChk( hipMalloc((int**)&count, (nPix)*sizeof(int)));
    gpuErrChk( hipMemset(count, 0, nPix*sizeof(int)));

    //Allocate memory for weights
    gpuErrChk( hipMalloc((unsigned int**)&wts, (nPix*numWtsBands)*sizeof(unsigned int)));
   gpuErrChk( hipMemset(wts, 0, nPix*numWtsBands*sizeof(unsigned int))  );

}

void gpuParams::setConstants()
{
    int constants[6];
    constants[0] = numCols;
    constants[1] = numLines;
    constants[2] = nBands;
    constants[3] = Nx;
    constants[4] = Ny;
    constants[5] = numWtsBands;

    gpuErrChk( hipMemcpyToSymbol(HIP_SYMBOL(d_inpts_int), constants, (6*sizeof(int))));

    //int readback[6];
    //gpuErrChk( hipMemcpyFromSymbol(readback, HIP_SYMBOL(d_inpts_int), (6*sizeof(int))));
    //std::cout << "Ncols = " << readback[0] << "\n"
    //          << "Nlines = " << readback[1] << "\n"
    //          << "Nbands = " << readback[2] << "\n"
    //          << "Nx = " << readback[3] << "\n"
    //          << "Ny = " << readback[4] << "\n"
    //          << "Nwts = " << readback[5] << "\n";

}

//Deallocate memory on GPU
void gpuParams::deallocateArrays()
{
    //Free amplitude
    gpuErrChk( hipFree(amplitude));

    //Free mask
    gpuErrChk( hipFree(mask));

    //Free output
    gpuErrChk( hipFree(count));

    //Free weights
    gpuErrChk( hipFree(wts));
}

//Pass amplitude and mask to GPU
void gpuParams::setInputs(float *amp,
                          unsigned char *msk)
{
    size_t nPix = numCols * numLines;
    
    //Copy amplitude to GPU
    gpuErrChk( hipMemcpy(amplitude, amp, (nPix*nBands*sizeof(float)),
            hipMemcpyHostToDevice));

    //Copy mask to GPU
    gpuErrChk( hipMemcpy(mask, msk, (nPix*sizeof(unsigned char)),
            hipMemcpyHostToDevice));
}

//Get count and wts from GPU
void gpuParams::getOutputs(int *cnt,
                           unsigned int *wmask)
{
    size_t nPix = numCols * numLines;

    //Copy count to host
    gpuErrChk( hipMemcpy(cnt, count, (nPix*sizeof(int)),
            hipMemcpyDeviceToHost));

    //copy wts to host
    gpuErrChk( hipMemcpy(wmask, wts, (nPix*numWtsBands*sizeof(unsigned int)), hipMemcpyDeviceToHost));

}

//Sort amplitudes
void gpuParams::sortAmplitude()
{
    int numPix = numCols * numLines;
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((numPix + (THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    /*if ((grid.x * THRD_PER_BLOCK) > numPix)
    {
        std::cout << " Number of empty threads = " << ((grid.x * THRD_PER_BLOCK) - numPix) << "\n";
    }*/

    runSortAmp <<<grid, block>>>(amplitude);

    //Track errors and synchronize
    gpuErrChk( hipGetLastError());
    gpuErrChk( hipDeviceSynchronize());
}

//Find neighbors
void gpuParams::process(double pval)
{
    //Copy the threshold value to device
    gpuErrChk( hipMemcpyToSymbol(HIP_SYMBOL(d_inpts_pval), &pval, sizeof(double)));

    int numPix = numCols * numLines;
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((numPix + (THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    /*if ((grid.x * THRD_PER_BLOCK) > numPix)
    {
        std::cout << "Number of empty threads = " << ((grid.x * THRD_PER_BLOCK) - numPix) << "\n";
    }*/

    findNeighbors <<<grid, block>>>(amplitude, mask,
                                    wts, count);

    //Track errors and synchronize
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());

}

/*****End of structure************/

/********** Actual Kernel function *************/
//This method is to sort a single pixel 
__global__ void runSortAmp(float *amp)
{
    //Pixel number 
    int pixel = (blockDim.x * blockIdx.x) + threadIdx.x;

    //Make sure count is within limits
    //i.e, pixel < numCols * numLines
    if (pixel < (d_inpts_int[0] * d_inpts_int[1]))
    {
        //Offset to pixel = pixel * nbands
        bruteForceSort(amp+(pixel*d_inpts_int[2])); 
    }
}


//This method is to identify neighbors for a single pixel
__global__ void findNeighbors(const float *amp, const unsigned char *mask,
                    unsigned int *wts, int *count)
{
    //Temporary variables needed
    int refii, refjj;
    int qq,ii,jj;
    double prob;
    const float *refpix;
    const float *cenpix;
    unsigned int *weight;

    //Pixel number
    int pp = (blockDim.x * blockIdx.x) + threadIdx.x;

    //Make sure count is within limits
    //i.e, pixel < numCols * numLines
    if (pp < (d_inpts_int[0] * d_inpts_int[1]))
    {
        if( mask[pp] != 0)
        {
            cenpix = amp + (d_inpts_int[2] * pp);
            weight = wts + (d_inpts_int[5] * pp);
            
            for(ii=-d_inpts_int[4]; ii<=d_inpts_int[4]; ii++)
            {
                refii = (pp/d_inpts_int[0]) + ii;

                for (jj=-d_inpts_int[3]; jj<=d_inpts_int[3]; jj++)
                {
                    refjj = (pp%d_inpts_int[0]) + jj;

                    if ((refii < d_inpts_int[1]) && (refii >=0) && 
                        (refjj < d_inpts_int[0]) && (refjj >=0)) 
                    {
                        qq = refii * d_inpts_int[0] + refjj;
                        refpix = amp + (d_inpts_int[2] * qq); 
                            
                        if (mask[qq] != 0)
                        {
                            //Count same pix as neighbor
                            if (pp == qq)
                            {
                                count[pp] += 1;
                                setBit(weight,
                                        0, 0,
                                        d_inpts_int[3],
                                        d_inpts_int[4],
                                        true);
                            }
                            else
                            {
                                prob = KS2test(cenpix, refpix,
                                                d_inpts_int[2]);
                                   
                                if (prob >= d_inpts_pval[0])
                                {
                                    count[pp] += 1;

                                    setBit( weight,
                                            ii,jj,
                                            d_inpts_int[3],
                                            d_inpts_int[4],
                                            true);
                                } //if prob > thresh
                            }  //if not same pixel
                        } //if ref pixel is not masked
                    } //if ref pixel is within limits
                } //loop over jj
            } //loop over ii 
            /*count[pp] = 0;
            for(ii=0; ii< d_inpts_int[2]; ii++)
                count[pp] += (cenpix[ii] == 0);*/

        } //if pixel is not masked
    } //if pixel is within limits
}

                
/********* End of actual kernel function **********/


//Actual interface to nmap.cpp
//This is the only function that is used by parent code directly.
void nmapProcessBlock(float *amp, unsigned char *msk,
                      int cols, int lines, int bands,
                      int *cnt, unsigned int *wmask,
                      int wtslen, double pval,
                      int Nx, int Ny)
{

    //Create structure to handle interaction with GPU
    struct gpuParams pars( cols,lines, bands,
                           Nx, Ny, wtslen);

    //Copy inputs to GPU
    pars.setInputs(amp, msk);

    //Sort amplitudes since stats is between histograms
    pars.sortAmplitude();

    //Process the block
    pars.process(pval);

    //Get outputs from GPU
    pars.getOutputs(cnt, wmask);

}

//Wrappers for GPU access
void lockGPU()
{
    getGPUDevice(0);
}

void unlockGPU()
{
    releaseGPUDevice();
}
